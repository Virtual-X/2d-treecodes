#include "hip/hip_runtime.h"
/*
 *  treecode.cpp
 *  Part of MRAG/2d-treecode-potential
 *
 *  Created and authored by Diego Rossinelli on 2015-09-25.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <omp.h>

#include <cstdio>
#include <cassert>

#include "cuda-common.h"
#include "force-kernels.h"
#include "upward.h"

#define ACCESS(x) __ldg(&(x)) 

namespace EvaluateForce
{
     struct SharedBuffers
    {
	realtype scratch[64];
	int buffered_e2ps[8];
	int stack[LMAX * 3];
    };

#ifndef NDEBUG
    __constant__ int nnodes;
#endif
    __constant__ Tree::Node * nodes;
    __constant__ realtype * expansions, *xdata, *ydata, *vdata;

    /*__global__ void  __launch_bounds__(128, 16)
    evaluate(const realtype * const xts, const realtype * const yts, const realtype thetasquared, 
	     realtype * const xforce, realtype * const yforce, const int ndst)
    {
	assert(blockDim.x == 32);
	
	const int tid = threadIdx.x;
	const bool master = tid == 0;
	
	const int gid = threadIdx.y + blockDim.y * blockIdx.x;

	if (gid >= ndst)
	    return;
	
	const realtype xt = xts[gid];
	const realtype yt = yts[gid];

	extern __shared__ SharedBuffers ary[];

	realtype * scratch = ary[threadIdx.y].scratch;

	int * stack = ary[threadIdx.y].stack;
	int * buffered_e2ps = ary[threadIdx.y].buffered_e2ps;
	int counter_e2ps = 0;
	
	int stackentry = 0, maxentry = 0;

	if (master)
	    stack[0] = 0;
	
	realtype result = 0;

	while(stackentry > -1)
	{
	    const int nodeid = stack[stackentry--];
	    assert(nodeid < nnodes);

	    const Tree::Node * node = nodes + nodeid;
	    const realtype nodemass = ACCESS(node->mass);
	
	    if (nodemass == 0)
	       	continue;

	    const realtype xcom = ACCESS(node->xcom);
	    const realtype ycom = ACCESS(node->ycom);
	    const realtype r = ACCESS(node->r);
	    const realtype rx = xt - xcom;
	    const realtype ry = yt - ycom;
	    const realtype r2 = rx * rx + ry * ry;

	    if (r * r < thetasquared * r2)
 	    {
		if (master)
		    buffered_e2ps[counter_e2ps] = nodeid;

		counter_e2ps++;

		if (counter_e2ps == 8)
		{
		    counter_e2ps = 0;

		    const int mynodeid = buffered_e2ps[tid / 4];
		    assert(mynodeid < nnodes);
	    
		    const Tree::Node * mynode = nodes + mynodeid;

		    result += force_e2p(ACCESS(mynode->mass), xt - ACCESS(mynode->xcom), yt - ACCESS(mynode->ycom), 
					expansions + ORDER * (0 + 2 * mynodeid), 
					expansions + ORDER * (1 + 2 * mynodeid), scratch);
		}
	    }
	    else 
	    {
		if (!node->state.innernode)
		{
		    const int s = node->s;

		    result += potential_p2p(&xdata[s], &ydata[s], &vdata[s], node->e - s, xt, yt);
		}
		else
		{
		    if (master)   
		    {
			const int childbase = ACCESS(node->state.childbase);

			for(int c = 0; c < 4; ++c) 
			    stack[++stackentry] = childbase + c;
		    }
		    else
			stackentry += 4;
			    
		    maxentry = max(maxentry, stackentry);
		    assert(maxentry < LMAX * 3);
		}
	    }
	}

  	if (tid / 4 < counter_e2ps)
	{
	    const int mynodeid = buffered_e2ps[tid / 4];
	    assert(mynodeid < nnodes);
	    
	    const Tree::Node * mynode = nodes + mynodeid;

	    result += force_e2p(ACCESS(mynode->mass), xt - ACCESS(mynode->xcom), yt - ACCESS(mynode->ycom), 
				expansions + ORDER * (0 + 2 * mynodeid), 
				expansions + ORDER * (1 + 2 * mynodeid), scratch);
	}

	result += __shfl_xor(result, 16 );
	result += __shfl_xor(result, 8 );
	result += __shfl_xor(result, 4 );
	result += __shfl_xor(result, 2 );
	result += __shfl_xor(result, 1 );

	if (master)
	    results[gid] = result;
	    }*/


}

void reference_evaluate(realtype * const xforce, realtype * const yforce, 
			const realtype xt, const realtype yt, realtype thetasquared);

using namespace EvaluateForce;
   
extern "C"
__attribute__ ((visibility ("default")))
void treecode_force_mrag_solve(const realtype theta,
			       const realtype * const xsrc,
			       const realtype * const ysrc,
			       const realtype * const vsrc,
			       const int nsrc,
			       const realtype * const x0s,
			       const realtype * const y0s,
			       const realtype * const hs,
			       const int nblocks,
			       realtype * const xforce,
			       realtype * const yforce)
{
    const realtype thetasquared = theta * theta;
    
    realtype * device_x0s, *device_y0s, *device_hs, *device_xforce, *device_yforce;
    
    const int ndst = nblocks * BLOCKSIZE * BLOCKSIZE;

    CUDA_CHECK(hipMalloc(&device_x0s, sizeof(realtype) * ndst));
    CUDA_CHECK(hipMalloc(&device_y0s, sizeof(realtype) * ndst));
    CUDA_CHECK(hipMalloc(&device_hs, sizeof(realtype) * ndst));

    CUDA_CHECK(hipMalloc(&device_xforce, sizeof(realtype) * ndst));
    CUDA_CHECK(hipMalloc(&device_yforce, sizeof(realtype) * ndst));
    
    CUDA_CHECK(hipMemcpyAsync(device_x0s, x0s, sizeof(realtype) * nblocks, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(device_y0s, y0s, sizeof(realtype) * nblocks, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(device_hs, hs, sizeof(realtype) * nblocks, hipMemcpyHostToDevice));
    
    Tree::build(xsrc, ysrc, vsrc, nsrc, 512);
    //CUDA_CHECK(hipDeviceSynchronize());
#if 0
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(xdata), &Tree::device_xdata, sizeof(Tree::device_xdata)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(ydata), &Tree::device_ydata, sizeof(Tree::device_ydata)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(vdata), &Tree::device_vdata, sizeof(Tree::device_vdata)));

    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(nodes), &Tree::device_nodes, sizeof(Tree::device_nodes)));
#ifndef NDEBUG
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(nnodes), &Tree::nnodes, sizeof(Tree::nnodes)));
#endif
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(expansions), &Tree::device_expansions, sizeof(Tree::device_expansions)));

    const int yblocksize = 4;
    evaluate<<<(ndst + yblocksize - 1) / yblocksize, dim3(32, yblocksize),
	sizeof(SharedBuffers) * yblocksize>>>(device_xdst, device_ydst, thetasquared, device_xforce, device_yforce, ndst);
    CUDA_CHECK(hipPeekAtLastError());
         
    CUDA_CHECK(hipMemcpyAsync(xdst, device_xforce, sizeof(realtype) * ndst, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpyAsync(ydst, device_yforce, sizeof(realtype) * ndst, hipMemcpyDeviceToHost));
#else
    for(int c = 0, b = 0; b < nblocks; ++b)
	for(int iy = 0; iy < BLOCKSIZE; ++iy)
	    for(int ix = 0; ix < BLOCKSIZE; ++ix, ++c)
	    {
		const realtype xdst = x0s[b] + hs[b] * ix;
		const realtype ydst = y0s[b] + hs[b] * iy;

		reference_evaluate(xforce + c, yforce + c, xdst, ydst, thetasquared);
	    }
#endif
    
    Tree::dispose();

    CUDA_CHECK(hipFree(device_x0s));
    CUDA_CHECK(hipFree(device_y0s));   
    CUDA_CHECK(hipFree(device_hs));
    CUDA_CHECK(hipFree(device_xforce));
    CUDA_CHECK(hipFree(device_yforce));
}

void reference_evaluate(realtype * const xforce, realtype * const yforce, 
			const realtype xt, const realtype yt, realtype thetasquared)
{
    const double eps = 10 * __DBL_EPSILON__;
    int stack[LMAX * 3];

    int stackentry = 0, maxentry = 0;
    
    stack[0] = 0;

    *xforce = 0;
    *yforce = 0;
	
    while(stackentry > -1)
    {
	const int nodeid = stack[stackentry--];
	    
	const Tree::Node * const node = Tree::host_nodes + nodeid;

	if (node->e - node->s == 0)
	    continue;
	    
	const realtype r2 = pow(xt - node->xcom, 2) + pow(yt - node->ycom, 2);

	if (node->r * node->r < thetasquared * r2)
	{
	    const realtype * rxp = Tree::host_expansions + ORDER * (0 + 2 * nodeid);
	    const realtype * ixp = Tree::host_expansions + ORDER * (1 + 2 * nodeid);

	    const realtype rz = xt - node->xcom;
	    const realtype iz = yt - node->ycom;

	    const realtype rinvz_1 = rz / r2;
	    const realtype iinvz_1 = -iz / r2;

	    realtype rsum = node->mass * rinvz_1, isum = node->mass * iinvz_1;
	    realtype rprod = rinvz_1, iprod = iinvz_1;

	    for(int j = 0; j < ORDER; ++j)
	    {
		const realtype rtmp = rprod * rinvz_1 - iprod * iinvz_1;
		const realtype itmp = rprod * iinvz_1 + iprod * rinvz_1;
		
		rprod = rtmp;
		iprod = itmp;	

		rsum -= (j + 1) * (rxp[j] * rprod - ixp[j] * iprod);
		isum -= (j + 1) * (rxp[j] * iprod + ixp[j] * rprod);
	    }
	    
	    *xforce += rsum;
	    *yforce += -isum;
	}
	else
	    if (!node->state.innernode)
	    {
		const int s = node->s;
		    
		realtype xsum = 0, ysum = 0;
		for(int i = s; i < node->e; ++i)
		{
		    const realtype xr = xt - Tree::host_xdata[i];
		    const realtype yr = yt - Tree::host_ydata[i];
		    const realtype factor = Tree::host_vdata[i] / (xr * xr + yr * yr + eps);

		    xsum += xr * factor;
		    ysum += yr * factor;
		}
    
		*xforce += xsum;
		*yforce += ysum;
	    }
	    else
	    {
		for(int c = 0; c < 4; ++c)
		    stack[++stackentry] = node->state.childbase + c;
		    
		if (maxentry < stackentry)
		    maxentry = stackentry;
	    }
    }
}

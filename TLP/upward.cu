#include "hip/hip_runtime.h"
/*
 *  treecode.cpp
 *  Part of MRAG/2d-treecode-potential
 *
 *  Created and authored by Diego Rossinelli on 2015-09-25.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cassert>

#include <omp.h>
#include <algorithm>
#include <limits>
#include <utility>

#include "upward.h"
#include "sort-sources.h"
#include "upward-kernels.h"
#include "cuda-common.h"

#define WARPSIZE 32
#define MFENCE
//__threadfence()
#define NQUEUES 4
#define LQSIZE 16

namespace Tree
{
    texture<int, hipTextureType1D> texKeys;

    __device__ int lower_bound(int s, int e, const int val)
    {
	int c = e - s;

	if (tex1Dfetch(texKeys, s) >= val)
	    return 0;

	if (tex1Dfetch(texKeys, e - 1) < val)
	    return e;

	while (c)
	{
	    int candidate_s = s, candidate_e = e;

	    const float h = (e - s) * 1.f/ WARPSIZE;
	    const int i = min(e - 1, (int)(s + threadIdx.x * h + 0.499999f));

	    const bool isless = tex1Dfetch(texKeys, i) < val;
	    candidate_s = isless ? i : s;
	    candidate_e = isless ? e : i;

#pragma unroll
	    for(int mask = WARPSIZE / 2 ; mask > 0 ; mask >>= 1)
	    {
		candidate_s = max(candidate_s, __shfl_xor(candidate_s, mask));
		candidate_e = min(candidate_e, __shfl_xor(candidate_e, mask));
	    }

	    s = candidate_s;
	    e = candidate_e;
	    c = min(c / 32, e - s);
	}

	return s + 1;
    }

    __device__ int upper_bound(int s, int e, const int val)
    {
	int c = e - s;

	if (tex1Dfetch(texKeys, s) > val)
	    return 0;

	if (tex1Dfetch(texKeys, e - 1) <= val)
	    return e;

	while (c)
	{
	    int candidate_s = s, candidate_e = e;

	    const float h = (e - s) * 1.f / WARPSIZE;
	    const int i = min(e - 1, (int)(s + threadIdx.x * h + 0.499999f));

	    const bool isless = tex1Dfetch(texKeys, i) <= val;
	    candidate_s = isless ? i : s;
	    candidate_e = isless ? e : i;

#pragma unroll
	    for(int mask = WARPSIZE / 2 ; mask > 0 ; mask >>= 1)
	    {
		candidate_s = max(candidate_s, __shfl_xor(candidate_s, mask));
		candidate_e = min(candidate_e, __shfl_xor(candidate_e, mask));
	    }

	    s = candidate_s;
	    e = candidate_e;
	    c = min(c / 32, e - s);
	}

	return s + 1;
    }

    __constant__ realtype *xsorted, *ysorted, *vsorted;

    struct NodeHelper
    {
	int x, y, l, mask, parent, validchildren;
	realtype w, wx, wy;

	__device__ void setup(int x, int y, int l, int mask, int parent)
	    {
		this->x = x;
		this->y = y;
		this->l = l;
	
		this->mask = mask;
		this->parent = parent;
		this->validchildren = 0;
	    }
    };

//TREE info
    __constant__ int bufsize;
    __device__ int currnnodes;
    
    __constant__ Node * bufnodes;
    __constant__ NodeHelper * bufhelpers;
    __constant__ realtype * bufexpansion;

    __device__ void process_leaf(const int nodeid, realtype extent)
    {
	const int tid = threadIdx.x;
	const bool master = tid == 0;

	Node * node = bufnodes + nodeid;
	NodeHelper * helper = bufhelpers + nodeid;

	const int s = node->s;
	const int e = node->e;

	realtype msum = 0, wsum = 0, wxsum = 0, wysum = 0;

	for(int t = s + tid; t < e; t += WARPSIZE)
	{
	    const realtype x = xsorted[t];
	    const realtype y = ysorted[t];
	    const realtype m = vsorted[t];
	    const realtype w = fabs(m);

	    msum += m;
	    wsum += w;
	    wxsum += x * w;
	    wysum += y * w;
	}

#pragma unroll
	for(int mask = WARPSIZE / 2 ; mask > 0 ; mask >>= 1)
	{
	    msum += __shfl_xor(msum, mask);
	    wsum += __shfl_xor(wsum, mask);
	    wxsum += __shfl_xor(wxsum, mask);
	    wysum += __shfl_xor(wysum, mask);
	}

	const realtype xcom = wsum ? wxsum / wsum : 0;
	const realtype ycom = wsum ? wysum / wsum : 0;

	upward_p2e(xcom, ycom,
		   xsorted + s, ysorted + s, vsorted + s, e - s,
		   bufexpansion + ORDER * (2 * nodeid + 0),
		   bufexpansion + ORDER * (2 * nodeid + 1));

	realtype r2 = 0;
	for(int i = s + tid; i < e; i += WARPSIZE)
	{
	    const realtype xr = xsorted[i] - xcom;
	    const realtype yr = ysorted[i] - ycom;

	    r2 = max(r2, xr * xr + yr * yr);
	}

#pragma unroll
	for(int mask = WARPSIZE / 2 ; mask > 0 ; mask >>= 1)
	{
	    const realtype other_r2 = __shfl_xor(r2, mask);
	    r2 = max(r2, other_r2);
	}

	if (master)
	{
	    helper->w = wsum;
	    helper->wx = wxsum;
	    helper->wy = wysum;

	    node->mass = msum;
	    node->r = sqrt(r2);
	    node->xcom = xcom;
	    node->ycom = ycom;
	    	    
	    MFENCE;
	}

	while(helper->parent >= 0)
	{
	    Node * parent = bufnodes + helper->parent;
	    NodeHelper * parenthelper = bufhelpers + helper->parent;
	    
	    bool e2e = false;

	    if (master)
		e2e = 3 == atomicAdd(&parenthelper->validchildren, 1);

	    e2e = __shfl(e2e, 0);

	    if (e2e)
	    {
		realtype xcom_parent, ycom_parent;

		if (master)
		{
		    realtype msum = 0, wsum = 0, wxsum = 0, wysum = 0;
		    for(int c = 0; c < 4; ++c)
		    {
			const int childid = parent->state.children[c];

			const Node * child = bufnodes + childid;
			msum += child->mass;

			const NodeHelper * childhelper = bufhelpers + childid;
			wsum += childhelper->w;
			wxsum += childhelper->wx;
			wysum += childhelper->wy;
		    }

		    parent->mass = msum;
		    parenthelper->w = wsum;
		    parenthelper->wx = wxsum;
		    parenthelper->wy = wysum;

		    assert(wsum);
		    xcom_parent = wxsum / wsum;
		    ycom_parent = wysum / wsum;

		    realtype rr = 0;
		    for(int c = 0; c < 4; ++c)
		    {
			const int childid = parent->state.children[c];
			const Node * child = bufnodes + childid;
			const NodeHelper * childhelper = bufhelpers + childid;

			if (childhelper->w)
			{
			    const realtype rx = xcom_parent - child->xcom;
			    const realtype ry = ycom_parent - child->ycom;

			    rr = max(rr, child->r + sqrt(rx * rx + ry * ry));
			}
		    }

		    parent->r = min(rr, 1.4143f * extent / (1 << parenthelper->l));
		    parent->xcom = xcom_parent;
		    parent->ycom = ycom_parent;
		}

		xcom_parent = __shfl(xcom_parent, 0);
		ycom_parent = __shfl(ycom_parent, 0);

		if (tid < 4)
		{
		    const int childid = parent->state.children[tid];
		    const Node * chd = bufnodes + childid;

		    upward_e2e(chd->xcom - xcom_parent, chd->ycom - ycom_parent, chd->mass,
			       bufexpansion + ORDER * (2 * childid + 0),
			       bufexpansion + ORDER * (2 * childid + 1),
			       bufexpansion + ORDER * (2 * helper->parent + 0),
			       bufexpansion + ORDER * (2 * helper->parent + 1));
		}

		if (master)
		    MFENCE;
	    }
	    else
		break;

	    node = parent;
	    helper = parenthelper;
	}
    }

//QUEUE info
    __constant__ int queuesize, * queues[NQUEUES];
    __device__  int qlock[NQUEUES], qhead[NQUEUES], qtail[NQUEUES], qtailnext[NQUEUES], qitems;
    __device__ bool qgood;

    __global__ void setup(const int nsrc)
    {
	currnnodes = 1;
	bufnodes[0].setup(0, nsrc);
	bufhelpers[0].setup(0, 0, 0, 0, -1);

	for(int i = 0; i < NQUEUES; ++i)
	{
	    qlock[i] = 1;
	    qhead[i] = 0;
	    qtail[i] = 0;
	    qtailnext[i] = 0;
	}

	const int qid = 0;
	queues[qid][0] = 0;
	qtail[qid] = 1;
	qtailnext[qid] = 1;

	qitems = 1;
	qgood = true;
    }

    __global__ void build_tree(const int LEAF_MAXCOUNT, const double extent)
    {
	assert(blockDim.x == warpSize && WARPSIZE == warpSize);

#if LQSIZE > 0
	__shared__ int ltasks[LQSIZE];

	{
	    const int tid2d = threadIdx.x + blockDim.x * threadIdx.y;

	    for(int i = tid2d; i < LQSIZE; i += blockDim.x * blockDim.y)
		ltasks[i] = -1;

	    __syncthreads();
	}
#endif

	const int tid = threadIdx.x;
	const bool master = tid == 0;

	int currid = -1;

	int iteration = -1;

	while(qitems && qgood)
	{
	    const int qid = (++iteration + blockIdx.x) % NQUEUES;

	    if (currid == -1)
	    {
		if (master)
		{
#if LQSIZE > 0
		    //get a task from the local pool if possible
		    for(int i = 0; i < LQSIZE && currid == -1; ++i)
			currid = atomicExch(ltasks + i, -1);
#endif

		    //or take one task from the global queues (if any)
		    if (currid == -1)
			if (atomicCAS(&qlock[qid], 1, 0)) 
			{
			    const int currhead = qhead[qid];

			    if (currhead < qtail[qid])
			    {
				const int entry = currhead % queuesize;

				currid = queues[qid][entry];

				qhead[qid] = currhead + 1;

				MFENCE;
			    }

			    qlock[qid] = 1;
			}
		}

		currid = __shfl(currid, 0);
	    }

	    if (currid >= 0)
	    {
		Node * node = bufnodes + currid;
		NodeHelper * helper = bufhelpers + currid;
		
		const int s = node->s;
		const int e = node->e;
		const int l = helper->l;

		const bool leaf = e - s <= LEAF_MAXCOUNT || l + 1 > LMAX;

		if (leaf)
		{
		    process_leaf(currid, extent);

		    if (master)
			atomicSub(&qitems, 1);

		    currid = -1;
		}
		else
		{
		    if (master) //children allocation
		    {
			const int bufbase = atomicAdd(&currnnodes, 4);

			if (bufbase + 4 > bufsize)
			{
			    qgood = false;
			    break;
			}

			for(int c = 0; c < 4; ++c)
			    node->state.children[c] = bufbase + c;
		    }

		    const int mask = helper->mask;
		    const int x = helper->x;
		    const int y = helper->y;

		    for(int c = 0; c < 4; ++c)
		    {
			const int shift = 2 * (LMAX - l - 1);

			const int key1 = mask | (c << shift);
			const int key2 = key1 + (1 << shift) - 1;

			const size_t indexmin = c == 0 ? s : lower_bound(s, e, key1);
			const size_t indexsup = c == 3 ? e : upper_bound(s, e, key2);

			if (master)
			{
			    const int childid = node->state.children[c];
			    Node * child = bufnodes + childid;
			    NodeHelper * childhelper = bufhelpers + childid;

			    child->setup(indexmin, indexsup);
			    childhelper->setup((x << 1) + (c & 1), (y << 1) + (c >> 1), l + 1, key1, currid);
			}
		    }

		    if (master) //enqueue new tasks
		    {
			bool placed_locally = false;

			const int localtask = node->state.children[2];
			
#if (LQSIZE > 0)
			//try to put a task in the local queue
			for(int i = 0; i < LQSIZE && !placed_locally; ++i)
			    placed_locally = atomicCAS(ltasks + i, -1, localtask) == -1;
#endif
			const int ngtasks = 3 - placed_locally;

			const int base = atomicAdd(&qtailnext[qid], ngtasks);

			if (base + ngtasks - qhead[qid] >= queuesize)
			{
			    qgood = false;
			    break;
			}
			else
			{
			    for(int c = 0; c < ngtasks; ++c)
				queues[qid][(base + c) % queuesize] = node->state.children[c];

			    atomicAdd(&qitems, 3);

			    MFENCE;

			    atomicAdd(&qtail[qid], ngtasks);
			}

			currid = node->state.children[3];
		    }
		}

		currid = __shfl(currid, 0);
	    }
	}
    }

    struct BuildResult
    {
	int ntreenodes, queuesize, nqueueitems;
	bool good;
    };

    __global__ void conclude(BuildResult * result)
    {
	result->ntreenodes = currnnodes;
	result->queuesize = qtail - qhead;
	result->nqueueitems = qitems;
	result->good = qgood;
    }

    realtype xmin, ymin, extent, *host_xdata, *host_ydata, *host_vdata;
    
    realtype * host_expansions = NULL, *device_expansions = NULL;
    Node * host_nodes = NULL;
    int nnodes = 0;
    
    realtype *device_xdata = NULL, *device_ydata = NULL, *device_vdata = NULL;
    Node * device_nodes = NULL;
    NodeHelper * device_helpers = NULL;
    int * device_keys = NULL;

    hipStream_t stream = 0;
    hipEvent_t evstart, evstop;

    int * device_queue;
    BuildResult * device_diag;
}

namespace TreeCheck
{
    void verify_all(const realtype * const xsrc, const realtype * const ysrc, const realtype * const vsrc, const int nsrc, const int LEAF_MAXCOUNT);
}

void Tree::build(const realtype * const xsrc,
		 const realtype * const ysrc,
		 const realtype * const vsrc,
		 const int nsrc,
		 const int LEAF_MAXCOUNT)
{
    //CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(build_tree), hipFuncCachePreferL1) );
    
    texKeys.channelDesc = hipCreateChannelDesc<int>();
    texKeys.filterMode = hipFilterModePoint;
    texKeys.mipmapFilterMode = hipFilterModePoint;
    texKeys.normalized = 0;

    CUDA_CHECK(hipEventCreate(&evstart));
    CUDA_CHECK(hipEventCreate(&evstop));
    
    int nsmxs = -1;
    CUDA_CHECK(hipDeviceGetAttribute (&nsmxs, hipDeviceAttributeMultiprocessorCount, 0));
    printf("i have found %d smxs\n", nsmxs);
   
    const int device_queuesize = 8e4;
    const int device_bufsize = 8e4;

    CUDA_CHECK(hipMalloc(&device_queue, sizeof(*device_queue) * device_queuesize * NQUEUES));
    CUDA_CHECK(hipMalloc(&device_nodes, sizeof(*device_nodes) * device_bufsize));
    CUDA_CHECK(hipMalloc(&device_helpers, sizeof(*device_helpers) * device_bufsize));    
    CUDA_CHECK(hipMalloc(&device_expansions, sizeof(realtype) * ORDER * 2 * device_bufsize));
    CUDA_CHECK(hipMalloc(&device_xdata, sizeof(realtype) * nsrc));
    CUDA_CHECK(hipMalloc(&device_ydata, sizeof(realtype) * nsrc));
    CUDA_CHECK(hipMalloc(&device_vdata, sizeof(realtype) * nsrc));
    CUDA_CHECK(hipMalloc(&device_keys, sizeof(int) * nsrc));
    
    CUDA_CHECK(hipHostMalloc(&device_diag, sizeof(*device_diag)));
   
    size_t textureoffset = 0;
    CUDA_CHECK(hipBindTexture(&textureoffset, &texKeys, device_keys, &texKeys.channelDesc, sizeof(int) * nsrc));
    assert(textureoffset == 0);

    CUDA_CHECK(hipPeekAtLastError());

    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(xsorted), &device_xdata, sizeof(device_xdata)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(ysorted), &device_ydata, sizeof(device_ydata)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(vsorted), &device_vdata, sizeof(device_vdata)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(bufsize), &device_bufsize, sizeof(device_bufsize)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(bufnodes), &device_nodes, sizeof(device_nodes)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(bufhelpers), &device_helpers, sizeof(device_helpers)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(bufexpansion), &device_expansions, sizeof(device_expansions)));
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(queuesize), &device_queuesize, sizeof(device_queuesize)));

    {
	int * ptrs[NQUEUES];
	for(int i = 0; i < NQUEUES; ++i)
	    ptrs[i] = device_queue + device_queuesize * i;

	CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(queues), &ptrs, sizeof(ptrs)));
    }

    CUDA_CHECK(hipMemcpyAsync(device_xdata, xsrc, sizeof(realtype) * nsrc, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(device_ydata, ysrc, sizeof(realtype) * nsrc, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(device_vdata, vsrc, sizeof(realtype) * nsrc, hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(evstart));
    
    sort_sources(stream, device_xdata, device_ydata, device_vdata, nsrc, device_keys, &xmin, &ymin, &extent);

    setup<<<1, 1>>>(nsrc);

    const int ysize = 16;
    build_tree<<<nsmxs * 2, dim3(32, ysize), sizeof(realtype) * 4 * 4 * ORDER * ysize>>>(LEAF_MAXCOUNT, extent);
    CUDA_CHECK(hipPeekAtLastError());

    conclude<<<1, 1>>>(device_diag);
    CUDA_CHECK(hipPeekAtLastError());

    CUDA_CHECK(hipEventRecord(evstop));
 
#ifndef NDEBUG
    TreeCheck::verify_all(xsrc, ysrc, vsrc, nsrc, LEAF_MAXCOUNT);
#endif
} 

void Tree::dispose()
{   
    CUDA_CHECK(hipEventSynchronize(evstop));

    float timems;
    CUDA_CHECK(hipEventElapsedTime(&timems, evstart,evstop  ));
    printf("\x1B[33mtimems: %f\x1b[0m\n", timems);

    printf("device has found %d nodes, and max queue size was %d, outstanding items %d, queue is good: %d\n",
	   device_diag->ntreenodes, device_diag->queuesize, device_diag->nqueueitems, device_diag->good);

    CUDA_CHECK(hipFree(device_xdata));
    CUDA_CHECK(hipFree(device_ydata));
    CUDA_CHECK(hipFree(device_vdata));
    CUDA_CHECK(hipFree(device_nodes));
    CUDA_CHECK(hipFree(device_expansions));
    CUDA_CHECK(hipFree(device_keys));
    CUDA_CHECK(hipFree(device_helpers));
    CUDA_CHECK(hipEventDestroy(evstart));
    CUDA_CHECK(hipEventDestroy(evstop));
    CUDA_CHECK(hipFree(device_queue));
    CUDA_CHECK(hipHostFree(device_diag));

#ifndef NDEBUG
    CUDA_CHECK(hipHostFree(host_nodes));
    CUDA_CHECK(hipHostFree(host_expansions));
    free(host_xdata);
    free(host_ydata);
    free(host_vdata);
#endif
}

namespace TreeCheck
{   
    int LEAF_MAXCOUNT;

    int * debug_keys = NULL;

    struct DebugNode
    {
	int x, y, l, s, e;
	bool leaf;
	realtype w, wx, wy, mass, r;

	DebugNode * children[4];

	void setup(int x, int y, int l, int s, int e, bool leaf)
	    {
		this->x = x;
		this->y = y;
		this->l = l;
		this->s = s;
		this->e = e;
		this->leaf = leaf;

	    }

	realtype xcom() const { return wx / w; }
	realtype ycom() const { return wy / w; }

	DebugNode() 
	    { 
		for (int i = 0; i < 4; ++i) 
		    children[i] = nullptr; 

		w = wx = wy = mass = r = 0; 
	    }
		
	typedef realtype alignedvec[ORDER] __attribute__ ((aligned (32)));

	alignedvec rexpansions;
	alignedvec iexpansions;

	void allocate_children()
	    {
		for(int i = 0; i < 4; ++i)
		    children[i] = new DebugNode;
	    }
	
	realtype * rexp(){return &rexpansions[0];} 
	realtype * iexp(){return &iexpansions[0];}
	
	void p2e(const realtype * __restrict__ const xsources,
		 const realtype * __restrict__ const ysources,
		 const realtype * __restrict__ const vsources,
		 const double x0, const double y0, const double h)
	    {
		reference_upward_p2e(xsources, ysources, vsources, e - s,
				     x0, y0, h, &mass, &w, &wx, &wy, &r,
				     rexpansions, iexpansions);
	    }
	void e2e()
	    {
		realtype srcmass[4], rx[4], ry[4];
		realtype * chldrxp[4], *chldixp[4];

		for(int c = 0; c < 4; ++c)
		{
		    DebugNode * chd = children[c];

		    srcmass[c] = chd->mass;
		    rx[c] = chd->xcom() - xcom();
		    ry[c] = chd->ycom() - ycom();
		    chldrxp[c] = chd->rexpansions;
		    chldixp[c] = chd->iexpansions;
		}

		reference_upward_e2e(rx, ry, srcmass, chldrxp, chldixp, rexpansions, iexpansions);
#ifndef NDEBUG
		{
		    for(int i = 0; i < ORDER; ++i)
			assert(!std::isnan((double)rexpansions[i]) && !std::isnan(iexpansions[i]));
		}
#endif
	    }

	~DebugNode() 
	    {
		for(int i = 0; i < 4; ++i)
		    if (children[i])
		    {
			delete children[i];

			children[i] = nullptr;
		    }
	    }
    };

    DebugNode * debugroot = NULL;

    void _build(DebugNode * const node, const int x, const int y, const int l, const int s, const int e, const int mask)
    {
	const double h = Tree::extent / (1 << l);
	const double x0 = Tree::xmin + h * x, y0 = Tree::ymin + h * y;

	assert(x < (1 << l) && y < (1 << l) && x >= 0 && y >= 0);
	//printf("node %d %d l%d\n", x, y, l);
	
	for(int i = s; i < e; ++i)
	    assert(Tree::host_xdata[i] >= x0 && Tree::host_xdata[i] < x0 + h && Tree::host_ydata[i] >= y0 && Tree::host_ydata[i] < y0 + h);
	
	node->setup(x, y, l, s, e, e - s <= LEAF_MAXCOUNT || l + 1 > LMAX);

	if (node->leaf)
	{
	    node->p2e(&Tree::host_xdata[s], &Tree::host_ydata[s], &Tree::host_vdata[s], x0, y0, h);

	    assert(node->r < 1.5 * h);
	}
	else
	{
	    node->allocate_children();

	    for(int c = 0; c < 4; ++c)
	    {
		const int shift = 2 * (LMAX - l - 1);

		const int key1 = mask | (c << shift);
		const int key2 = key1 + (1 << shift) - 1;

		const size_t indexmin = c == 0 ? s : std::lower_bound(debug_keys + s, debug_keys + e, key1) - debug_keys;
		const size_t indexsup = c == 3 ? e : std::upper_bound(debug_keys + s, debug_keys + e, key2) - debug_keys;

		DebugNode * chd = node->children[c];

		_build(chd, (x << 1) + (c & 1), (y << 1) + (c >> 1), l + 1, indexmin, indexsup, key1);
	    }

	    for(int c = 0; c < 4; ++c)
	    {
		DebugNode * chd = node->children[c];
		node->mass += chd->mass;
		node->w += chd->w;
		node->wx += chd->wx;
		node->wy += chd->wy;

		node->children[c] = chd;
	    }

	    node->r = 0;
	    for(int c = 0; c < 4; ++c)
		if (node->children[c]->w)
		    node->r = std::max(node->r,
				       node->children[c]->r +
				       sqrt(pow(node->xcom() - node->children[c]->xcom(), 2) +
					    pow(node->ycom() - node->children[c]->ycom(), 2)));

	    node->r = std::min(node->r, 1.4143 * h);

	    assert(node->r < 1.5 * h);

	    {
		realtype r = 0;

		for(int i = s; i < e; ++i)
		    r = std::max(r, pow(Tree::host_xdata[i] - node->xcom(), (realtype)2) + pow(Tree::host_ydata[i] - node->ycom(), (realtype)2));

		assert (sqrt(r) <= node->r);
	    }

	    node->e2e();
	}

	assert(node->xcom() >= x0 && node->xcom() < x0 + h && node->ycom() >= y0 && node->ycom() < y0 + h || node->e - node->s == 0);
    }

    bool verbose = false;

    int check_bits(double x, double y)
    {
	union ASD
	{
	    unsigned char c[8];
	    double d;
	};

	ASD a, b;
	a.d = x;
	b.d = y;

	int currbit = 0;
	for(int i = 0; i < 8; ++i)
	{
	    unsigned char c1 = a.c[7 - i], c2 = b.c[7 - i];

	    for(int b = 0; b < 8; ++b, ++currbit)
	    {
		if (((c1 >> b) & 1) != ((c2 >> b) & 1))
		{
		    if (verbose) printf("numbers differ from the %d most-significant bit\n", currbit);
		    return currbit;
		}
	    }
	}

	return currbit;
    }

    int check_bits(const double *a , const double *b, const int n)
    {
	if (verbose) printf("*******************************\n");
	int r = 64;

	for(int i = 0; i < n; ++i)
	{
	    if (fabs(a[i]) > 1.11e-16 || fabs(b[i]) > 1.11e-16)
	    {
		int l = check_bits(a[i], b[i]);
		const double x = a[i];
		const double y = b[i];
		if (l < 48 )
		    if (verbose) printf("strange case of %+.20e vs %+.20e relerr %e\n", x, y, (x - y) / y);
		r= min(r, l );
	    }
	}

	if (verbose) printf("********** end ***************\n");
	return r;
    }

    void check_tree (const int EXPORD, const int nodeid, realtype * allexp, Tree::Node * allnodes, Tree::Node& a, TreeCheck::DebugNode& b)
    {
	//assert(a.x == b.x);
	//assert(a.y == b.y);
	//assert(a.l == b.l);
	assert(a.s == b.s);
	assert(a.e == b.e);
	//assert(a.mask == b.mask);
	if (verbose)
	{
	    printf("<%s>", (b.leaf ? "LEAF" : "INNER"));
	    printf("ASDnode %d %d l%d s: %d e: %d. check passed..\n", b.x, b.y, b.l, b.s, b.e);
/*
  printf("a/ m-w-wx-wy-r: %.20e %.20e %.20e %.20e %.20e\n",
  a.mass, a.w, a.wx, a.wy, a.r);
  printf("b/ m-w-wx-wy-r: %.20e %.20e %.20e %.20e %.20e\n",
  b.mass, b.w, b.wx, b.wy, b.r);*/
	}
	assert(check_bits(a.mass, b.mass) >= 40);
	//assert(check_bits(a.w, b.w) >= 40);
	assert(check_bits(a.xcom, b.wx / b.w) >= 40 || b.w == 0);
	assert(check_bits(a.ycom, b.wy / b.w) >= 40 || b.w == 0);
	
	//assert(check_bits(a.wx, b.wx) >= 40 || a.w == 0);
	//assert(check_bits(a.wy, b.wy) >= 40 || a.w == 0);

	assert(check_bits(a.r, b.r) >= 32);
#ifndef NDEBUG
	{
	    const realtype * resrexp = allexp + EXPORD * (2 * nodeid + 0);
	    const realtype * resiexp = allexp + EXPORD * (2 * nodeid + 1);
	    const realtype * refrexp = b.rexp();
	    const realtype * refiexp = b.iexp();
	    assert(24 <= check_bits(resrexp, refrexp, EXPORD));
	    assert(24 <= check_bits(resiexp, refiexp, EXPORD));
	}
#endif

	if (!b.leaf)
	    for(int c = 0; c < 4; ++c)
		check_tree(EXPORD, a.state.children[c], allexp, allnodes, allnodes[a.state.children[c]], *b.children[c]);
    }

    void verify_all(const realtype * const xsrc, const realtype * const ysrc, const realtype * const vsrc, const int nsrc, const int LEAF_MAXCOUNT)
    {
	CUDA_CHECK(hipStreamSynchronize(0));
	
	Tree::nnodes = Tree::device_diag->ntreenodes;
	const size_t expansionsbytes = sizeof(realtype) * 2 * ORDER * Tree::nnodes;
	
    
	posix_memalign((void **)&Tree::host_xdata, 32, sizeof(realtype) * nsrc);
	posix_memalign((void **)&Tree::host_ydata, 32, sizeof(realtype) * nsrc);
	posix_memalign((void **)&Tree::host_vdata, 32, sizeof(realtype) * nsrc);
	
	CUDA_CHECK(hipMemcpy(Tree::host_xdata, Tree::device_xdata, sizeof(realtype) * nsrc, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(Tree::host_ydata, Tree::device_ydata, sizeof(realtype) * nsrc, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(Tree::host_vdata, Tree::device_vdata, sizeof(realtype) * nsrc, hipMemcpyDeviceToHost));
	
	CUDA_CHECK(hipHostMalloc(&Tree::host_expansions, expansionsbytes));
	CUDA_CHECK(hipHostMalloc(&Tree::host_nodes, sizeof(Tree::Node) * Tree::nnodes));
	CUDA_CHECK(hipMemcpy(Tree::host_expansions, Tree::device_expansions, expansionsbytes, hipMemcpyDeviceToHost));

	std::vector<Tree::NodeHelper> devhelpers(Tree::nnodes);
	CUDA_CHECK(hipMemcpy(Tree::host_nodes, Tree::device_nodes, sizeof(Tree::Node) * Tree::nnodes, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&devhelpers.front(), Tree::device_helpers, sizeof(Tree::NodeHelper) * Tree::nnodes,
			      hipMemcpyDeviceToHost));
    
	printf("VERIFICATION _______________________________________\n");
	
	TreeCheck::LEAF_MAXCOUNT = LEAF_MAXCOUNT;

       	CUDA_CHECK(hipHostMalloc(&debug_keys, sizeof(int) * nsrc));

	CUDA_CHECK(hipMemcpy(debug_keys, Tree::device_keys, sizeof(int) * nsrc, hipMemcpyDeviceToHost));
	
	std::pair<int, int> * kv = NULL;
	posix_memalign((void **)&kv, 32, sizeof(*kv) * nsrc);

	//assert(truexmin == *std::min_element(xsrc, xsrc + nsrc));
	//assert(trueymin == *std::min_element(ysrc, ysrc + nsrc));

	//assert(ext0 == *std::max_element(xsrc, xsrc + nsrc) - truexmin);
	//assert(ext1 == *std::max_element(ysrc, ysrc + nsrc) - trueymin);

	for(int i = 0; i < nsrc; ++i)
	{
	    int x = floor((xsrc[i] - Tree::xmin) / Tree::extent * (1 << LMAX));
	    int y = floor((ysrc[i] - Tree::ymin) / Tree::extent * (1 << LMAX));

	    assert(x >= 0 && y >= 0);
	    assert(x < (1 << LMAX) && y < (1 << LMAX));

	    x = (x | (x << 8)) & 0x00FF00FF;
	    x = (x | (x << 4)) & 0x0F0F0F0F;
	    x = (x | (x << 2)) & 0x33333333;
	    x = (x | (x << 1)) & 0x55555555;

	    y = (y | (y << 8)) & 0x00FF00FF;
	    y = (y | (y << 4)) & 0x0F0F0F0F;
	    y = (y | (y << 2)) & 0x33333333;
	    y = (y | (y << 1)) & 0x55555555;

	    const int key = x | (y << 1);

	    kv[i].first = key;
	    kv[i].second = i;
	}

	std::sort(kv, kv + nsrc);

	for(int i = 0; i < nsrc; ++i)
	{
	    //const int key = kv[i].first;

	    const int entry = kv[i].second;
	    assert(entry >= 0 && entry < nsrc);

	    assert(Tree::host_xdata[i] == xsrc[entry]);
	    assert(Tree::host_ydata[i] == ysrc[entry]);
	    assert(Tree::host_vdata[i] == vsrc[entry]);
	}

	printf("SORTING IS GOOD\n");
	
	free(kv);

	debugroot = new DebugNode;
	
	_build(debugroot, 0, 0, 0, 0, nsrc, 0);
    
	//const int nnodes = Tree::nnodes;
	//std::vector<Tree::Node> allnodes(nnodes);

	//CUDA_CHECK(hipMemcpy(&allnodes.front(), Tree::device_bufnodes, sizeof(Tree::DeviceNode) * allnodes.size(), hipMemcpyDeviceToHost));

	//std::vector<realtype> allexpansions(nnodes * ORDER * 2);
	//CUDA_CHECK(hipMemcpy(&allexpansions.front(), device_bufexpansions, sizeof(realtype) * 2 * ORDER * nnodes, hipMemcpyDeviceToHost));

	printf("rooot xylsem: %d %d, children %d %d %d %d\n",
	       Tree::host_nodes[0].s, Tree::host_nodes[0].e,
	       Tree::host_nodes[0].state.children[0],
	       Tree::host_nodes[0].state.children[1],
	       Tree::host_nodes[0].state.children[2],
	       Tree::host_nodes[0].state.children[3]);

	//ok let's check this
	check_tree(ORDER, 0, Tree::host_expansions,Tree::host_nodes, Tree::host_nodes[0], *debugroot);

	printf("TREE IS GOOD\n");
	printf("VERIFICATION SUCCEDED.______________________________\n");
	CUDA_CHECK(hipHostFree(debug_keys));
    }
}
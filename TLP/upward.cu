#include "hip/hip_runtime.h"
/*
 *  treecode.cpp
 *  Part of MRAG/2d-treecode-potential
 *
 *  Created and authored by Diego Rossinelli on 2015-09-25.
 *  Copyright 2015. All rights reserved.
 *
 *  Users are NOT authorized
 *  to employ the present software for their own publications
 *  before getting a written permission from the author of this file.
 */

#include <cassert>
//#include <cmath>

#include <omp.h>
//#include <parallel/algorithm>
#include <algorithm>
#include <limits>
#include <utility>

#include "upward.h"
#include "upward-kernels.h"
#include "cuda-common.h"

#define  _INSTRUMENTATION_
#ifndef _INSTRUMENTATION_
#define MYRDTSC 0
#else
#define MYRDTSC _rdtsc()
#endif

#define LMAX 15

namespace Tree
{
    int LEAF_MAXCOUNT;

    realtype ext, xmin, ymin;

    int * keys = NULL;

    realtype *xdata = NULL, *ydata = NULL, *vdata = NULL;

    Node * root = NULL;

    void _build(Node * const node, const int x, const int y, const int l, const int s, const int e, const int mask)
    {
	const int64_t startallc = MYRDTSC;

	const double h = ext / (1 << l);
	const double x0 = xmin + h * x, y0 = ymin + h * y;

	assert(x < (1 << l) && y < (1 << l) && x >= 0 && y >= 0);

#ifndef NDEBUG
	for(int i = s; i < e; ++i)
	    assert(xdata[i] >= x0 && xdata[i] < x0 + h && ydata[i] >= y0 && ydata[i] < y0 + h);
#endif

	node->setup(x, y, l, s, e, e - s <= LEAF_MAXCOUNT || l + 1 > LMAX);

	if (node->leaf)
	{
	    const int64_t startc = MYRDTSC;
	    node->p2e(&xdata[s], &ydata[s], &vdata[s], x0, y0, h);
	    node->p2ecycles = MYRDTSC - startc;

	    assert(node->r < 1.5 * h);
	}
	else
	{
	    node->allocate_children();

	    for(int c = 0; c < 4; ++c)
	    {
		const int shift = 2 * (LMAX - l - 1);

		const int key1 = mask | (c << shift);
		const int key2 = key1 + (1 << shift) - 1;

		const int64_t startc = MYRDTSC;
		const size_t indexmin = c == 0 ? s : std::lower_bound(keys + s, keys + e, key1) - keys;
		const size_t indexsup = c == 3 ? e : std::upper_bound(keys + s, keys + e, key2) - keys;
		node->searchcycles += MYRDTSC - startc;

		Node * chd = node->children[c];

#pragma omp task firstprivate(chd, c, x, y, l, indexmin, indexsup, key1) if (indexsup - indexmin > 5e3 && c < 3)
		//if (c < 3 && l < 8)
		{
		    _build(chd, (x << 1) + (c & 1), (y << 1) + (c >> 1), l + 1, indexmin, indexsup, key1);
		}

	    }
//#pragma omp taskyield
#pragma omp taskwait

	    const int64_t startc = MYRDTSC;

	    for(int c = 0; c < 4; ++c)
	    {
		Node * chd = node->children[c];
		node->mass += chd->mass;
		node->w += chd->w;
		node->wx += chd->wx;
		node->wy += chd->wy;

		node->children[c] = chd;
	    }

	    //realtype rcandidates[4];
	    node->r = 0;
	    for(int c = 0; c < 4; ++c)
		node->r = std::max(node->r,
				   node->children[c]->r +
				   sqrt(pow(node->xcom() - node->children[c]->xcom(), 2) +
					pow(node->ycom() - node->children[c]->ycom(), 2)));

	    node->r = std::min(node->r, 1.4143 * h);

	    assert(node->r < 1.5 * h);

#ifndef NDEBUG
	    {
		realtype r = 0;

		for(int i = s; i < e; ++i)
		    r = std::max(r, pow(xdata[i] - node->xcom(), (realtype)2) + pow(ydata[i] - node->ycom(), (realtype)2));

		assert (sqrt(r) <= node->r);
	    }
#endif

	    node->e2e();
	    node->e2ecycles = MYRDTSC - startc;
	}

#ifndef NDEBUG
	{
	    assert(node->xcom() >= x0 && node->xcom() < x0 + h && node->ycom() >= y0 && node->ycom() < y0 + h || node->e - node->s == 0);
	}
#endif

	const int64_t endallc = MYRDTSC;
	node->allcycles = endallc - startallc;
    }

 template <class ForwardIterator, class T>
 __device__ ForwardIterator lower_bound (ForwardIterator first, ForwardIterator last, const T& val)
{
  ForwardIterator it;
  int count, step;
  count = last - first; //distance(first,last);
  while (count>0)
  {
      
      it = first; step=count/2; it += step; //advance (it,step);
      // printf("step: %d\n", step);
    if (*it<val) {                 // or: if (comp(*it,val)), for version (2)
      first=++it;
      count-=step+1;
    }
    else count=step;
  }
  return first;
}

    template <class ForwardIterator, class T>
 __device__ ForwardIterator upper_bound (ForwardIterator first, ForwardIterator last, const T& val)
{
  ForwardIterator it;
  int count, step;
  count = last - first;//std::distance(first,last);
  while (count>0)
  {
      it = first; step=count/2; it += step;//std::advance (it,step);
    if (!(val<*it))                 // or: if (!comp(val,*it)), for version (2)
      { first=++it; count-=step+1;  }
    else count=step;
  }
  return first;
}
    
    
    __global__ void generate_keys(const realtype * const xsrc, const realtype * const ysrc, const int n,
				  const realtype xmin, const realtype ymin, const realtype ext,
				  int * const keys)
    {
	const int gid = threadIdx.x + blockDim.x * blockIdx.x;

	if (gid >= n)
	    return;

	int x = floor((xsrc[gid] - xmin) / ext * (1 << LMAX));
	int y = floor((ysrc[gid] - ymin) / ext * (1 << LMAX));
	
	assert(x >= 0 && y >= 0);
	assert(x < (1 << LMAX) && y < (1 << LMAX));
	
	x = (x | (x << 8)) & 0x00FF00FF;
	x = (x | (x << 4)) & 0x0F0F0F0F;
	x = (x | (x << 2)) & 0x33333333;
	x = (x | (x << 1)) & 0x55555555;
	
	y = (y | (y << 8)) & 0x00FF00FF;
	y = (y | (y << 4)) & 0x0F0F0F0F;
	y = (y | (y << 2)) & 0x33333333;
	y = (y | (y << 1)) & 0x55555555;
	
	const int key = x | (y << 1);
	
	keys[gid] = key;
    }


    //Node * const node;//, const int x, const int y, const int l, const int s, const int e, const int mask)
    __constant__ int * sorted_keys;
   
    struct DeviceNode
    {
	int x, y, l, s, e, mask;

	DeviceNode * children[4];

	__host__ __device__ void setup(int x, int y, int l, int s, int e, int mask)
	    {
		this->x = x;
		this->y = y;
		this->l = l;
		this->s = s;
		this->e = e;
		this->mask = mask;
		
		for (int i = 0; i < 4; ++i) 
		    children[i] = nullptr; 
	    }
	
	__device__ void allocate_children()
	    {
		for(int i = 0; i < 4; ++i)
		    children[i] = new DeviceNode;
	    }

	__device__ ~DeviceNode()
	    {
		for(int i = 0; i < 4; ++i)
		    if (children[i])
			delete children[i];
	    }
    };


#define QSIZE 1000
    __device__ DeviceNode * queue[QSIZE];
    __device__ int qlock, qhead, qtail, qtailnext, qitems;
    __device__ bool qgood;

    __global__ void place_root(DeviceNode * root, const int nsrc)
    {
	root->setup(0, 0, 0, 0, nsrc, false);

	queue[0] = root;
	
	qlock = 0;
	qhead = 0;
	qtail = 1;
	qtailnext = 1;
	qitems = 1;
	qgood = true;
    }

    __device__ DeviceNode * bcast_ptr(DeviceNode * ptr)
    {
	union Pack
	{
	    DeviceNode * ptr;
	    int words[2];
	};

	Pack p;
	p.ptr = ptr;

	const int w0 = __shfl(p.words[0], 0);
	const int w1 = __shfl(p.words[1], 0);
	
	p.words[0] = w0;
	p.words[1] = w1;

	return p.ptr;
    }
    
    __global__ void build_tree(const int LEAF_MAXCOUNT, int * kk)
    {	
	const int tid = threadIdx.x;
	const int slot = threadIdx.y;
	const bool master = tid == 0;
	
	DeviceNode * curr;

	while(qitems && qgood) 
	{
	    curr = NULL;
	    
	    if (master)
		if (0 == atomicCAS(&qlock, 0, 1))
		{
		    const int currhead = qhead;
		    
		    if (currhead < qtail)
		    {
			//printf("block %d slot %d got something \n", blockIdx.x, slot);
			
			const int entry = currhead % QSIZE;
			
			curr = queue[entry];

			qhead = currhead + 1;
		
			__threadfence();			
		    }

		    qlock = 0;
		}

	    curr = bcast_ptr(curr);
	    
	    if (curr && master)
	    {

		const int s = curr->s;
		const int e = curr->e;
		const int l = curr->l;
		
		const bool leaf = e - s <= LEAF_MAXCOUNT || l + 1 > LMAX;
		
		if (leaf)
		{
		    //compute P2E here
		    atomicSub(&qitems, 1);
		}		
		else
		{
		    curr->allocate_children();

		    const int mask = curr->mask;
		    const int x = curr->x;
		    const int y = curr->y;
		
		    for(int c = 0; c < 4; ++c)
		    {
			const int shift = 2 * (LMAX - l - 1);
			
			const int key1 = mask | (c << shift);
			const int key2 = key1 + (1 << shift) - 1;

			//printf("lowerbound: %d %d %d and ptr %p  -> %p\n", s, e, key1, sorted_keys, kk);

			const size_t indexmin = c == 0 ? s :  lower_bound(sorted_keys + s, sorted_keys + e, key1) - sorted_keys;
			const size_t indexsup = c == 3 ? e :  upper_bound(sorted_keys + s, sorted_keys + e, key2) - sorted_keys;
			
			curr->children[c]->setup((x << 1) + (c & 1), (y << 1) + (c >> 1), l + 1, indexmin, indexsup, key1);
		    }

		    const int base = atomicAdd(&qtailnext, 4);
		    //printf("base: %d\n", base);

		    if (base + 4 - qhead >= QSIZE)
		    {
			//printf("oooops base: %d, qhead: %d -> size %d\n", base, qhead, base - qhead);
			qgood = false;
		    }
		    else
		    {
			for(int c = 0; c < 4; ++c)
			    queue[(base + c) % QSIZE] = curr->children[c];

			atomicAdd(&qitems, 3);
			
			__threadfence();

			atomicAdd(&qtail, 4);
		    }
		}
	    }
	}

	assert(qgood);
    }
}

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/pair.h>
#include <thrust/sort.h>

void Tree::build(const realtype * const xsrc, const realtype * const ysrc, const realtype * const vsrc, const int nsrc,
		 Node * const root, const int LEAF_MAXCOUNT)
{
    Tree::LEAF_MAXCOUNT = LEAF_MAXCOUNT;
    
    posix_memalign((void **)&xdata, 32, sizeof(*xdata) * nsrc);
    posix_memalign((void **)&ydata, 32, sizeof(*ydata) * nsrc);
    posix_memalign((void **)&vdata, 32, sizeof(*vdata) * nsrc);
    posix_memalign((void **)&keys, 32, sizeof(int) * nsrc);
    
    realtype *device_xdata, *device_ydata, *device_vdata;
    CUDA_CHECK(hipDeviceReset());
    CUDA_CHECK(hipMalloc(&device_xdata, sizeof(realtype) * nsrc));
    CUDA_CHECK(hipMalloc(&device_ydata, sizeof(realtype) * nsrc));
    CUDA_CHECK(hipMalloc(&device_vdata, sizeof(realtype) * nsrc));

    DeviceNode * device_root;
    CUDA_CHECK(hipMalloc(&device_root, sizeof(*device_root)));

    int * device_keys;
    CUDA_CHECK(hipMalloc(&device_keys, sizeof(int) * nsrc));
    
#ifndef NDEBUG
    CUDA_CHECK(hipMemset(device_keys, 0xff, sizeof(int) * nsrc));
#endif

    CUDA_CHECK(hipMemcpy(device_xdata, xsrc, sizeof(realtype) * nsrc, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(device_ydata, ysrc, sizeof(realtype) * nsrc, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(device_vdata, vsrc, sizeof(realtype) * nsrc, hipMemcpyHostToDevice));

    thrust::pair<thrust::device_ptr<realtype>, thrust::device_ptr<realtype> > xminmax =
	thrust::minmax_element(thrust::device_pointer_cast(device_xdata), thrust::device_pointer_cast(device_xdata)  + nsrc);
    
    thrust::pair<thrust::device_ptr<realtype>, thrust::device_ptr<realtype> > yminmax =
	thrust::minmax_element(thrust::device_pointer_cast(device_ydata), thrust::device_pointer_cast(device_ydata)  + nsrc);
    
    const realtype truexmin = *xminmax.first;
    const realtype trueymin = *yminmax.first;
    
    const realtype ext0 = *xminmax.second - truexmin;
    const realtype ext1 = *yminmax.second - trueymin;

    const realtype eps = 10000 * std::numeric_limits<realtype>::epsilon();
    
    ext = std::max(ext0, ext1) * (1 + 2 * eps);
    xmin = truexmin - eps * ext;
    ymin = trueymin - eps * ext;

    generate_keys<<< (nsrc + 127)/128, 128>>>(device_xdata, device_ydata, nsrc,
					      xmin, ymin, ext, device_keys);

    CUDA_CHECK(hipPeekAtLastError());

    thrust::sort_by_key(thrust::device_pointer_cast(device_keys),
			thrust::device_pointer_cast(device_keys + nsrc),
			thrust::make_zip_iterator(thrust::make_tuple(
						      thrust::device_pointer_cast(device_xdata),
						      thrust::device_pointer_cast(device_ydata),
						      thrust::device_pointer_cast(device_vdata)))); 

    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(sorted_keys), &device_keys, sizeof(device_keys)));
    
    place_root<<<1, 1>>>(device_root, nsrc);
    build_tree<<<14 * 16, dim3(32, 4)>>>(LEAF_MAXCOUNT, device_keys);

    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    CUDA_CHECK(hipMemcpy(xdata, device_xdata, sizeof(realtype) * nsrc, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(ydata, device_ydata, sizeof(realtype) * nsrc, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(vdata, device_vdata, sizeof(realtype) * nsrc, hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipMemcpy(keys, device_keys, sizeof(int) * nsrc, hipMemcpyDeviceToHost));

    printf("bye!\n");
    exit(0);
	
#ifndef NDEBUG
    std::pair<int, int> * kv = NULL;
    
    posix_memalign((void **)&kv, 32, sizeof(*kv) * nsrc);

    assert(truexmin == *std::min_element(xsrc, xsrc + nsrc));
    assert(trueymin == *std::min_element(ysrc, ysrc + nsrc));
    
    assert(ext0 == *std::max_element(xsrc, xsrc + nsrc) - truexmin);
    assert(ext1 == *std::max_element(ysrc, ysrc + nsrc) - trueymin);
    
    for(int i = 0; i < nsrc; ++i)
    {
	int x = floor((xsrc[i] - xmin) / ext * (1 << LMAX));
	int y = floor((ysrc[i] - ymin) / ext * (1 << LMAX));
	
	assert(x >= 0 && y >= 0);
	assert(x < (1 << LMAX) && y < (1 << LMAX));
	
	x = (x | (x << 8)) & 0x00FF00FF;
	x = (x | (x << 4)) & 0x0F0F0F0F;
	x = (x | (x << 2)) & 0x33333333;
	x = (x | (x << 1)) & 0x55555555;
	
	y = (y | (y << 8)) & 0x00FF00FF;
	y = (y | (y << 4)) & 0x0F0F0F0F;
	y = (y | (y << 2)) & 0x33333333;
	y = (y | (y << 1)) & 0x55555555;
	
	const int key = x | (y << 1);
	
	kv[i].first = key;
	kv[i].second = i;
    }

    std::sort(kv, kv + nsrc);
   
    for(int i = 0; i < nsrc; ++i)
    {
	//const int key = kv[i].first;
	
	const int entry = kv[i].second;
	assert(entry >= 0 && entry < nsrc);
	
	assert(xdata[i] == xsrc[entry]);
	assert(ydata[i] == ysrc[entry]);
	assert(vdata[i] == vsrc[entry]);
    }

    free(kv);
#endif
  
#pragma omp parallel //num_threads(24)
    {
#pragma omp single
	{ _build(root, 0, 0, 0, 0, nsrc, 0); }
    }

    CUDA_CHECK(hipFree(device_xdata));
    CUDA_CHECK(hipFree(device_ydata));
    CUDA_CHECK(hipFree(device_vdata));
    CUDA_CHECK(hipFree(device_keys));
    CUDA_CHECK(hipFree(device_root));
}

void Tree::dispose()
{
    free(xdata);
    free(ydata);
    free(vdata);
    free(keys);
}
